#include "hip/hip_runtime.h"
/* Short explanation on the way to add fields and terms in this file
 *
 * evolver is a class that merely calls updates on all fields and terms
 * the arguments on its constructor are 
 *
 *      evolver system(x,           sx,             sy,             dx,       dy,       dt);
 *                     Use CUDA | x-system size | y-system size | delta_x | delta_y | delta_t
 *
 * To this evolver we can add fields:
 *
 *      system.createField( name, dynamic );
 *
 * name is a string and dynamic if a boolean that sets whether the field
 * is set in each step through a time derivative or through an equality.
 *
 * To each field we can add terms
 *      
 *      system.createTerm(  field_name, prefactor, {field_1, ..., field_n}  );
 *
 *  This term would be a term of "field_name", with that prefactor, that multiplies
 *  fields field_1 to field_n.
 */ 

#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <ostream>
#include "../inc/defines.h"
#include "../inc/evolver.h"
#include "../inc/field.h"
#include "../inc/term.h"

#ifdef WITHCUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif

#define NX 400
#define NY 400

void set_to_zero(float2 *, int, int);
__global__ void set_to_zero_k(float2*, int, int);

int main(int argc, char **argv)
{
    evolver system(1, NX, NY, 1.0f, 1.0f, 1.0f, 2);

    system.createField("phi", true);        // 0
    float D = 1.0f;

    system.fields[0]->isNoisy = true;
    system.fields[0]->noiseType = GaussianWhite;
    system.fields[0]->noise_amplitude = {D,0, 0, 0,0};

    hipMemcpy(system.fields[0]->real_array_d, system.fields[0]->real_array, NX*NY*sizeof(float2), hipMemcpyHostToDevice);
    hipMemcpy(system.fields[0]->comp_array_d, system.fields[0]->comp_array, NX*NY*sizeof(float2), hipMemcpyHostToDevice);

    system.fields[0]->toComp();

    for (int i = 0; i < system.fields.size(); i++)
    {
        system.fields[i]->prepareDevice();
        system.fields[i]->precalculateImplicit(system.dt);
    }
    system.fields[0]->outputToFile = true;

    int steps = 10000;
    int freq = 100;
    int check = steps/100;
    if (check < 1) check = 1;
    
    system.printInformation();

    for (int i = 0; i < steps; i++)
    {
        system.advanceTime();
        // do stuff with noise
        // apply filters here to cuda pointer:
        // system.fields[0]->comp_array_d
        // run system.fields[0]->toReal() to transform to real space
        // set phi to 0
        if (i%freq==0)
        {
            set_to_zero(system.fields[0]->real_array_d,NX,NY);
            system.fields[0]->toComp();
        }
        // output will always happen after setting to 0
        if (i % check == 0)
        {
            std::cout << "Progress: " << i/check << "%\r";
            std::cout.flush();
        }
    }

    return 0;
}


void set_to_zero(float2 *real_array, int sx, int sy)
{
    dim3 TPB(32,32);
    dim3 blocks(sx/32,sy/32);

    set_to_zero_k<<<blocks, TPB>>>(real_array, sx, sy);
}

__global__ void set_to_zero_k(float2 *real_array, int sx, int sy)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = j*sx + i;

    if (index < sx*sy)
    {
        real_array[index].x = 0.0f;
    }
}
